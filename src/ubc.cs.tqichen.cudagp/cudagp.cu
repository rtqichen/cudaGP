/*
 * Wraps the calls to kernels here.
 */

#include "cudagp.h"
#include <stdio.h>
#include "impl/kernels.h"

void randomFloats(float* h_matrix, int size) {
    for (int i=0; i<size; i++) {
        h_matrix[i] = rand() / (float)RAND_MAX * 100;
    }
}

void printMatrix(float* matrix, int numRows, int numCols) {
    printf("Printing %d by %d matrix:\n", numRows, numCols);
    for (int i=0; i<numRows; i++) {
        for (int j=0; j<numCols; j++) {
            printf("%.2f ", matrix[i*numCols+j]);
        }
        printf("\n");
    }
}

void readData(float* X, float* y, int n) {

    FILE *infile = fopen("testdata/grayroos.dat", "r");
    if (!infile) {
        printf("Failed to read file.");
    }

    int i=0;
    char line[100];
    int a,b;
    while(i<n && fgets(line, sizeof(line), infile) != NULL) {
        sscanf(line, "%d\t%d[^\n]", &a, &b);
        X[i] = (float) a;
        y[i] = (float) b;
        i++;
    }
}

float* constructCovMatrix_ref(float* X, int n, int d, Kernel_t kernel_string, float* h_kernel_params) {

    float* h_cov = (float*)malloc(n*n*sizeof(float));

    kernelfunc kernfunc = getKernelFunction(kernel_string);

    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            h_cov[i*n+j] = kernfunc(&X[i*d], &X[j*d], d, h_kernel_params);
        }
    }

    return h_cov;
}

//int main(int argc, const char** argv) {
//
//    srand(0);
//
//    float *X,*y;
//
//    int n=10000; // for full GP, this number <= 10^4
//    int d=50;
//    X = (float*)malloc(n*d*sizeof(float));
//    y = (float*)malloc(n*sizeof(float));
//    randomFloats(X,n*d);
//    randomFloats(y,n);
//
//    float params[1] = {1.0f};
//
//    // initialize the GP
//    cudagp_handle_t gp = initializeCudaGP(X,y,n,d, cudagpSquaredExponentialKernel, params);
//
//    printf("Done?\n");
//    hipDeviceSynchronize();
//    freeCudaGP(gp);
//    printf("Done!\n");
//
//    hipDeviceReset();
//    return EXIT_SUCCESS;
//}
