#include "hip/hip_runtime.h"
/*
 * Wraps the calls to kernels here.
 */

#include "cudagp.h"
#include <stdio.h>
#include <math.h>
#include "impl/kernels.h"
#include <time.h>
#include "impl/utils_cuda.h"

void randomFloats(float* h_matrix, int size) {
    for (int i=0; i<size; i++) {
        h_matrix[i] = rand() / (float)RAND_MAX * 100;
    }
}

void printMatrix(float* matrix, int numRows, int numCols) {
    printf("Printing %d by %d matrix:\n", numRows, numCols);
    for (int i=0; i<numRows; i++) {
        for (int j=0; j<numCols; j++) {
            printf("%.8f ", matrix[i*numCols+j]);
        }
        printf("\n");
    }
}

void printDiagOfMatrix(float* matrix, int numRows, int numCols) {
    printf("Printing diagonals of %d by %d matrix:\n", numRows, numCols);
    for (int i=0; i<min(numRows,numCols); i++) {
        printf("%.4f\n", matrix[i*numCols+i]);
    }
}

int readData(float* X, float* y, int n) {

    FILE *infile = fopen("test_data", "r");
    if (!infile) {
        printf("Failed to read file.");
    }

    int i=0;
    char line[100];
    float a,b;
    while(i<n && fgets(line, sizeof(line), infile) != NULL) {
        sscanf(line, "%f\t%f[^\n]", &a, &b);
        X[i] = a;
        y[i] = b;
        i++;
    }

    return n;
}

float* constructCovMatrix_ref(float* X, int n, int d, kernelstring_enum kernel_string, float* h_kernel_params) {

    float* h_cov = (float*)malloc(n*n*sizeof(float));

    kernelfunc kernfunc = getKernelFunction(kernel_string);

    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            h_cov[i*n+j] = kernfunc(&X[i*d], &X[j*d], d, h_kernel_params);
        }
    }

    return h_cov;
}

float* linspace(int min, int max, int len) {
    float *x = (float*) malloc(len*sizeof(float));
    for (int i=0; i<len; i++) {
        x[i] = min + (max-min)*(i/(float)(len-1));
    }
    return x;
}

/**
 * Generates random numbers uniformly in [min, max]
 */
float* uniform(int len, float max, float min) {
    float *x = (float*) malloc(len*sizeof(float));
    for (int i=0; i<len; i++) {
        x[i] = ((float)rand()/(float)RAND_MAX) * (max-min+1) + min;
    }
    return x;
}

float* func(float *x, int n) {
    float *y = (float*) malloc(n*sizeof(float));
    for (int i=0; i<n; i++) {
        y[i] = sin(x[i]) + ((float)rand()/(float)RAND_MAX) * 0.4;
    }
    return y;
}

double getMean(double *x, int size) {
    double sum = 0.0;
    for (int i=0; i<size; i++) {
        sum += x[i];
    }
    return sum/size;
}

double getStdev(double *x, int size) {
    double var = 0.0;
    double mean = getMean(x, size);
    for (int i=0; i<size; i++) {
        var += (x[i]-mean)*(x[i]-mean);
    }
    return sqrt(var/(size-1));
}

void warmup() {
    int n = 30;
    int d = 1;

    float *X = uniform(n, -400, 400);
    float *y = func(X, n);

    int t = 201;
    float* Xtest = linspace(-400, 400, t);

    float params[2] = {1.8, 1.15};

    cudagphandle_t cudagphandle = initializeCudaGP(X,y,n,d, cudagpSquaredExponentialKernel, params);
    prediction_t pred = predict(cudagphandle, Xtest, t);

    free(Xtest);
    free(X);
    free(y);
    freeCudaGP(cudagphandle);
    free(pred.mean);
    free(pred.var);
}

#define NTESTS 8

int main(int argc, const char** argv) {

    srand(0);

    int n = 7000;
    int d = 1;
    //float* X = (float*)malloc(n*d*sizeof(float));
    //float* y = (float*)malloc(n*sizeof(float));
    //readData(X, y, n);

    float *X = uniform(n, -400, 400);
    float *y = func(X, n);

    warmup();

    printf("Using N=%d training data.\n", n);

//    printf("Data:\n");
//    printMatrix(X, n, 1);
//    printMatrix(y, n, 1);

    int t = 201;
    float* Xtest = linspace(-400, 400, t);

    float params[2] = {1.8, 1.15};

    int ntrials = 20;

    // time the full GP
//    do {
//
//        printf("Timing the Full GP implementation . . .\n") ;
//        clock_t tic = clock();
//        for (int i=0; i<ntrials; i++) {
//            cudagphandle_t cudagphandle = initializeCudaGP(X,y,n,d, cudagpSquaredExponentialKernel, params);
//            prediction_t pred = predict(cudagphandle, Xtest, t);
//        }
//        clock_t toc = clock();
//        printf("Full GP Prediction - Elapsed time: %f seconds\n\n", (double)(toc - tic) / CLOCKS_PER_SEC / ntrials);
//
//    } while (false);

    // time the clustered GP with k clusters
    do {
        printf("Timing the data-parallel GP implementation . . .\n");
        int numClusters[NTESTS] = {1,10,20,50,100,200,500,1000};
        double times[ntrials];
        for (int k=0; k<NTESTS; k++) {

            clock_t tic;
            clock_t toc;
            for (int i=0; i<ntrials; i++) {
                tic = clock();
                cudagphandle_t cudagphandle2 = initializeCudaDGP(X,y,n,d, cudagpSquaredExponentialKernel, numClusters[k], params);
                prediction_t pred2 = predict(cudagphandle2, Xtest, t);
                toc = clock();
                times[i] = (double)(toc - tic) / CLOCKS_PER_SEC;

                freeCudaGP(cudagphandle2);
                free(pred2.mean);
                free(pred2.var);
            }
            double mean = getMean(times,ntrials);
            double stdev = getStdev(times,ntrials);
            printf("K=%d Sparse GP Prediction - Elapsed time: %f seconds with std %f seconds\n", numClusters[k], mean, stdev);
        }
    } while (false);

//    printf("Xtest:\n"); printMatrix(Xtest, t, 1);
//    printf("Mean:\n"); printMatrix(pred2.mean, t, 1);
//    printf("Marginal Variance:\n"); printMatrix(pred2.var, t, 1);

//    free(X);
//    free(y);
//    free(Xtest);
//
//    freeCudaGP(cudagphandle);
//    freeCudaGP(cudagphandle2);
//
//    free(pred.mean);
//    free(pred.var);
//    free(pred2.mean);
//    free(pred2.var);
//
//    printf("Done!\n");
//
//    hipDeviceReset();
    return EXIT_SUCCESS;
}
