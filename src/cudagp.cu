#include "hip/hip_runtime.h"
/*
 * Wraps the calls to kernels here.
 */

#include "cudagp.h"
#include <stdio.h>
#include "impl/kernels.h"

void randomFloats(float* h_matrix, int size) {
    for (int i=0; i<size; i++) {
        h_matrix[i] = rand() / (float)RAND_MAX * 100;
    }
}

void printMatrix(float* matrix, int numRows, int numCols) {
    printf("Printing %d by %d matrix:\n", numRows, numCols);
    for (int i=0; i<numRows; i++) {
        for (int j=0; j<numCols; j++) {
            printf("%.4f ", matrix[i*numCols+j]);
        }
        printf("\n");
    }
}

int readData(float* X, float* y, int n) {

    FILE *infile = fopen("testdata/grayroos.dat", "r");
    if (!infile) {
        printf("Failed to read file.");
    }

    int i=0;
    char line[100];
    int a,b;
    while(i<n && fgets(line, sizeof(line), infile) != NULL) {
        sscanf(line, "%d\t%d[^\n]", &a, &b);
        X[i] = (float) a;
        y[i] = (float) b;
        i++;
    }

    return n;
}

float* constructCovMatrix_ref(float* X, int n, int d, Kernel_t kernel_string, float* h_kernel_params) {

    float* h_cov = (float*)malloc(n*n*sizeof(float));

    kernelfunc kernfunc = getKernelFunction(kernel_string);

    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            h_cov[i*n+j] = kernfunc(&X[i*d], &X[j*d], d, h_kernel_params);
        }
    }

    return h_cov;
}

float* linspace(int min, int max, int len) {
    float *x = (float*) malloc(len*sizeof(float));
    for (int i=0; i<len; i++) {
        x[i] = min + (max-min)*(i/(float)(len-1));
    }
    return x;
}

int main(int argc, const char** argv) {

    srand(0);

    int n = 42;
    int d = 1;
    float* X = (float*)malloc(n*d*sizeof(float));
    float* y = (float*)malloc(n*sizeof(float));
    readData(X, y, n);

    int t = 201;
    float* Xtest = linspace(500, 860, t);

    float params[1] = {8.0f};

    // initialize the GP
    cudagphandle_t cudagphandle = initializeCudaGP(X,y,n,d, cudagpSquaredExponentialKernel, params);

    prediction_t pred = predict(cudagphandle, Xtest, t);

    printMatrix(pred.mean, t, 1);

    freeCudaGP(cudagphandle);
    printf("Done!\n");

    hipDeviceReset();
    return EXIT_SUCCESS;
}
