#include <stdio.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "../cudagp.h"
#include "cov.h"
#include "utils_cuda.h"

/*
 * Initializes the sparseGP by assigning data points to separate experts.
 *
 * Specifically, the initialization function performs the following tasks:
 *
 * (1) The full covariance matrix is computed.
 * (2) Data points are clustered (?).
 * (3) Each cluster is assigned to an expert.
 */

/*
 * Copies the dataset (X,y) onto the GPU
 */
dataset_t transferDataToDevice(const float *h_X, const float* h_y, const int n, const int d) {

    float *d_X, *d_y;
    checkCudaErrors(hipMalloc((void**)&d_X,sizeof(float)*n*d));
    checkCudaErrors(hipMalloc((void**)&d_y,sizeof(float)*n));

    checkCudaErrors(hipMemcpyAsync(d_X,h_X,sizeof(float)*n*d,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_y,h_y,sizeof(float)*n,hipMemcpyHostToDevice));

    dataset_t d_data;
    d_data.X = d_X;
    d_data.y = d_y;
    d_data.d = d;
    d_data.n = n;

    return d_data;
}

/**
 * Transfers the full dataset onto the GPU.
 * Then calculates the covariance matrix along with its Cholesky decomposition.
 */
cudagphandle_t initializeCudaGP(
        const float *h_X,
        const float* h_y,
        const int n,
        const int d,
        const Kernel_t kernel,
        float* h_defaultParams,
        bool useDefaultparams) {

    // --- Transfer dataset to GPU
    dataset_t d_ds = transferDataToDevice(h_X,h_y,n,d);

    // --- Set kernel parameters
    int np = numParams(kernel);
    float* h_params;

    if (!h_defaultParams) {
        h_params = (float*) malloc(np*sizeof(float));
        for (int i=0; i<np; i++) {
            h_params[i] = rand() / RAND_MAX; // is there a smarter way to do initialization? this can introduce numerical instability.
        }
    } else {
        h_params = h_defaultParams;
    }

    float* d_params;
    checkCudaErrors(hipMalloc((void**)&d_params, np*sizeof(float)));
    checkCudaErrors(hipMemcpy(d_params, h_params, np*sizeof(float), hipMemcpyHostToDevice));

    // --- CuBLAS initialization
    hipblasHandle_t cublashandle;
    hipblasCreate(&cublashandle);

    // --- CuSolver initialization
    hipsolverHandle_t cusolverhandle;
    hipsolverDnCreate(&cusolverhandle);

    // --- CudaGP handle
    cudagphandle_t cudagphandle;
    cudagphandle.d_dataset = d_ds;
    cudagphandle.kernel = kernel;
    cudagphandle.numParams = np;
    cudagphandle.d_params = d_params;
    cudagphandle.cusolverHandle = cusolverhandle;
    cudagphandle.cublasHandle = cublashandle;

    return cudagphandle;
}

cudagphandle_t initializeCudaGP(float *h_X, float* h_y, int n, int d, Kernel_t kernel) {
    return initializeCudaGP(h_X, h_y, n, d, kernel, 0, false);
}

cudagphandle_t initializeCudaGP(float *h_X, float* h_y, int n, int d, Kernel_t kernel, float* defaultparams) {
    return initializeCudaGP(h_X, h_y, n, d, kernel, defaultparams, true);
}

void freeCudaGP(cudagphandle_t ahandle) {
    checkCudaErrors(hipFree(ahandle.d_dataset.X));
    checkCudaErrors(hipFree(ahandle.d_dataset.y));
    checkCudaErrors(hipFree(ahandle.d_params));
    checkCusolverErrors(hipsolverDnDestroy(ahandle.cusolverHandle));
    hipblasDestroy(ahandle.cublasHandle);
}
